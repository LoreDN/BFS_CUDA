
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sys/time.h>
#include <time.h>
#include <vector>

#define MAX_FRONTIER_SIZE 128
#define DIM_GRID 1024
#define DIM_BLOCK 128

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

void read_matrix(std::vector<int> &row_ptr,
                 std::vector<int> &col_ind,
                 std::vector<float> &values,
                 const std::string &filename,
                 int &num_rows,
                 int &num_cols,
                 int &num_vals);

__device__ void insertIntoFrontier(int val, int *frontier, int *frontier_size) {
  frontier[*frontier_size] = val;
  *frontier_size           = *frontier_size + 1;
}

__device__ inline void swap(int **ptr1, int **ptr2) {
  int *tmp = *ptr1;
  *ptr1    = *ptr2;
  *ptr2    = tmp;
}


// BFS algorithm optimized for GPU
__global__ void BFS_gpu(const int *source_ptr, const int *rowPointers, const int *destinations, int *distances)
{
  // initialize frontiers
  __shared__ int currentFrontier[MAX_FRONTIER_SIZE];
  __shared__ int currentFrontierSize;
  __shared__ int previousFrontier[MAX_FRONTIER_SIZE];
  __shared__ int previousFrontierSize;
  
  // initialize block's previous frontier from source
  if (threadIdx.x == 0)
  {
    currentFrontierSize = 0;
    previousFrontierSize = 0;
    const int source = *source_ptr;
    insertIntoFrontier(source, previousFrontier, &previousFrontierSize);
    distances[source] = 0;
  }

  __syncthreads();

  // BFS with parallel vertices
  while(previousFrontierSize > 0)       // while there are new vertices to visit
  {
    // visit all vertices on the previus frontier
    if(blockIdx.x < previousFrontierSize)
    {
      int currentVertex = previousFrontier[blockIdx.x];
      int row_start = rowPointers[currentVertex];
      int row_end = rowPointers[currentVertex + 1];

      // check all outgoing edges
      for(int row_i = row_start + threadIdx.x; row_i < row_end; row_i += DIM_BLOCK)      // parallelize over all outgoing edges even if they are more than the block size
      {
        if(distances[destinations[row_i]] == -1)
        {
          // this vertex has not been visited yet
          insertIntoFrontier(destinations[row_i], currentFrontier, &currentFrontierSize);
          distances[destinations[row_i]] = distances[currentVertex] +1;
        }
      }
    }
    
    // wait for all vertices to be visited
    __syncthreads();
    
    // swap to the next frontier
    if(threadIdx.x == 0)
    {
      swap((int**)&currentFrontier, (int**)&previousFrontier);
      previousFrontierSize = currentFrontierSize;
      currentFrontierSize  = 0;
    }

    // synchronize with the swap
    __syncthreads();

  }
}


int main(int argc, char *argv[]) {
  if (argc != 3) {
    printf("Usage: ./exec matrix_file source\n");
    return 0;
  }

  // host variables allocation
  std::vector<int> host_row_ptr;
  std::vector<int> host_col_ind;
  std::vector<float> values;
  int num_rows, num_cols, num_vals;

  const std::string filename{argv[1]};
  // The node starts from 1 but array starts from 0
  const int host_source = atoi(argv[2]) - 1;

  read_matrix(host_row_ptr, host_col_ind, values, filename, num_rows, num_cols, num_vals);

  // Initialize dist to -1
  std::vector<int> host_dist(num_vals);
  for (int i = 0; i < num_vals; i++) { host_dist[i] = -1; }

  // gpu variables allocation
  int *gpu_row_ptr;
  int *gpu_col_ind;
  int *gpu_dist;
  int *gpu_source;

  // gpu memory allocation
  CHECK(hipMalloc(&gpu_source, sizeof(int)));
  CHECK(hipMalloc(&gpu_row_ptr, host_row_ptr.size() * sizeof(int)));
  CHECK(hipMalloc(&gpu_col_ind, host_col_ind.size() * sizeof(int)));
  CHECK(hipMalloc(&gpu_dist, num_vals * sizeof(int)));

  // Copy data from host to device
  CHECK(hipMemcpy(gpu_source, &host_source, sizeof(int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gpu_row_ptr, host_row_ptr.data(), host_row_ptr.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gpu_col_ind, host_col_ind.data(), host_col_ind.size() * sizeof(int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gpu_dist, host_dist.data(), num_vals * sizeof(int), hipMemcpyHostToDevice));

  // Call the gpu_kernel function
  BFS_gpu<<<DIM_GRID,DIM_BLOCK>>>(gpu_source, gpu_row_ptr, gpu_col_ind, gpu_dist);
  CHECK_KERNELCALL();

  // gpu memory free
  CHECK(hipFree(gpu_source));
  CHECK(hipFree(gpu_row_ptr));
  CHECK(hipFree(gpu_col_ind));
  CHECK(hipFree(gpu_dist));
  
  return EXIT_SUCCESS;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(std::vector<int> &row_ptr,
                 std::vector<int> &col_ind,
                 std::vector<float> &values,
                 const std::string &filename,
                 int &num_rows,
                 int &num_cols,
                 int &num_vals) {
  std::ifstream file(filename);
  if (!file.is_open()) {
    std::cerr << "File cannot be opened!\n";
    throw std::runtime_error("File cannot be opened");
  }

  // Get number of rows, columns, and non-zero values
  file >> num_rows >> num_cols >> num_vals;

  row_ptr.resize(num_rows + 1);
  col_ind.resize(num_vals);
  values.resize(num_vals);

  // Collect occurrences of each row for determining the indices of row_ptr
  std::vector<int> row_occurrences(num_rows, 0);

  int row, column;
  float value;
  while (file >> row >> column >> value) {
    // Subtract 1 from row and column indices to match C format
    row--;
    column--;

    row_occurrences[row]++;
  }

  // Set row_ptr
  int index = 0;
  for (int i = 0; i < num_rows; i++) {
    row_ptr[i] = index;
    index += row_occurrences[i];
  }
  row_ptr[num_rows] = num_vals;

  // Reset the file stream to read again from the beginning
  file.clear();
  file.seekg(0, std::ios::beg);

  // Read the first line again to skip it
  file >> num_rows >> num_cols >> num_vals;

  std::fill(col_ind.begin(), col_ind.end(), -1);

  int i = 0;
  while (file >> row >> column >> value) {
    row--;
    column--;

    // Find the correct index (i + row_ptr[row]) using both row information and an index i
    while (col_ind[i + row_ptr[row]] != -1) { i++; }
    col_ind[i + row_ptr[row]] = column;
    values[i + row_ptr[row]]  = value;
    i                         = 0;
  }
}
